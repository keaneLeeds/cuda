#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>

#define N 512
__global__ void add(int *a, int *b, int *c) {
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
	//putc(a[threadIdx.x], fp);
	//putc(' ', fp);
	//putc(b[threadIdx.x], fp);
	//putc(' ', fp);
	//putc(c[threadIdx.x], fp);
	//putc(',', fp);
	//putc(' ', fp);  
}

void random_ints(int *a, int size) {
	int i;
	time_t t;	
	srand((unsigned) time(&t));
	for (i=0; i<size; i++) {
		a[i] = 1; //rand() % 64;
		if(i == size-1) {
			a[i] = 2;
		}
		//printf("i: %d\na[i]: %d\n", i, a[i]);
	}
}

void zeroarr(int *a, int size) {
	for (int i=0; i < size; i++) {
		a[i] = 0;
	}
}

void printarr(int *a, int size) {
	for(int i=0 ; i < size; i++) {
		printf("%d", a[i]);
	}
	printf("\n\n");
}
 
int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	//d*;
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	//FILE *fp;
	//fp = fopen("dump.txt", "a");
	//int *d_d;
	int size = N * sizeof(int);

	// Allocate space for device copies of a, b, c
	checkCudaErrors(hipMalloc((void **) &d_a, size));
	checkCudaErrors(hipMalloc((void **) &d_b, size));
	checkCudaErrors(hipMalloc((void **) &d_c, size));

	//hipMalloc((void**) &d_d, size*6);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *) malloc(size);
	random_ints(a, N);
	printf("A before sendng to device\n");
	printarr(a, N);

	b = (int *) malloc(size);
	printf("B before sending to device\n");
	random_ints(b, N);
	printarr(b, N);

	c = (int *) malloc(size);
	printf("C before sending to device\n");
	zeroarr(c, N);
	printarr(c, N);

	//d = (int *) malloc(size*6);
	//printf("D before sending to device\n");
	//zeroarr(d, N*6);
	//printarr(d, N*6);

	//printf("a: %d\nb: %d\nc: %d\n", *a, *b, *c);

	// Copy inputs to device
	//printf("d_a: %d, a %d", *d_a, *a);
	checkCudaErrors(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));

	// Launch add() kernel on GPU
	add<<<1,N>>>(d_a, d_b, d_c);

	// copy result back to host
	checkCudaErrors(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost));
	printf("D_C from device\n");
	printarr(c, N);

	//hipMemcpy(d, d_d, size*6, hipMemcpyDeviceToHost);
	//printf("D_D from device\n");
	//printarr(d, size*6);

	printf("TOTAL: %d\n", *c);

	// Cleanup
	free(a);
	hipFree(d_a);
	free(b);
	hipFree(d_b);
	free(c);
	hipFree(d_c);
	return 0;
}


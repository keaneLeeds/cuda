
#include <hip/hip_runtime.h>
#import <stdio.h>

__global__ void add(int *a, int *b, int *c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n) {
		c[index] = a[index] + b[index];
	}
}

void random_ints(int *a, int N) {
	int i;
	for (i=0; i<N; ++i) {
		a[i] = rand();
	}
}

#define N (2048*2048)
#define M 512
int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // devic copies of a, b, c
	int size = N * sizeof(int);

	// Allocate space for device copies of a, b, c
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *) malloc(size); random_ints(a, N);
	b = (int *) malloc(size); random_ints(b, N);
	c = (int *) malloc(size);

	// copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<(N+M-1)/M,M>>>(d_a, d_b, d_c, N);

	// copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// Cleanup
	free(a);
	hipFree(d_a);
	free(b);
	hipFree(d_b);
	free(c);
	hipFree(d_c);
	return 0;
}

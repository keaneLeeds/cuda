
#include <hip/hip_runtime.h>
#import <stdio.h>

__global__ void add(int *a, int *b, int*c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int *a, int N) {
	int i;
	for (i=0; i<N; ++i) {
		a[i] = rand();
	}
}

#define N 512
int main(int argc, char *argv[]) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // devic copies of a, b, c
	int size = N * sizeof(int);

	// Allocate space for device copies of a, b, c
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *) malloc(size); random_ints(a, N);
	b = (int *) malloc(size); random_ints(b, N);
	c = (int *) malloc(size);
	printf("a: %d\nb: %d\nc: %d\n", a, b, c);

	// copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<N,1>>>(d_a, d_b, d_c);

	// copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// Cleanup
	free(a);
	hipFree(d_a);
	free(b);
	hipFree(d_b);
	free(c);
	hipFree(d_c);
	return 0;
}
